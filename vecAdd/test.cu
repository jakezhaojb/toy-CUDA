#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "gputimer.h"

#define checkCudaError(err) { __checkCudaError((err), __FILE__, __LINE__); }

void __checkCudaError(hipError_t err, const char* file, int line){
    if(err != hipSuccess){
        fprintf(stderr, "%s(%i), CUDA RuntimeError %d, %s\n", file, line, int(err), hipGetErrorString(err));    
        exit(-1);
    }        
}


__global__ void vecAddKernel(float* A, float* B, float* C, int n){
    //int i = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.x;
    if(i < n){
        C[i] = A[i] + B[i];
    }
}


void vecAdd(float* A, float* B, float* C, int n){
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    vecAddKernel<<<1, 256>>>(d_A, d_B, d_C, n);
    checkCudaError( hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost) );
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, const char *argv[])
{
    GpuTimer timer;
    float A[10] = {1,2,3,4,5,6,7,8,9,10};
    float B[10] = {10,20,30,40,50,60,70,80,90,100};
    //float* C = malloc(10 * sizeof(float));
    float C[10] = {0};
    timer.Start();
    vecAdd(A, B, C, 10);
    timer.Stop();
    for (int i = 0; i < 10; i++) {
        printf("%f, ", C[i]);
    }
    printf("\n");
    printf("Time elapsed: %g ms \n", timer.Elapsed());
    return 0;
}
